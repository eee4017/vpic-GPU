#include "hip/hip_runtime.h"
#define IN_spa

#include <hipcub/hipcub.hpp>

#include "advance_p_gpu.cuh"
#include "gpu_util.cuh"
#include "move_p_gpu.cuh"

#define SHARE_MAX_VOXEL_SIZE 2  // 18

__global__ void handle_particle_movers(advance_p_gpu_args args, int temp_nm) {
  const int block_rank = blockIdx.x;
  const int n_block = gridDim.x;
  const int thread_rank = threadIdx.x;
  const int n_thread = blockDim.x;
  const int stride_size = args.stride_size;

  const float qsp = args.qsp;
  int itmp, n;
  int *nm = args.nm;

  GPU_DISTRIBUTE(temp_nm, stride_size, block_rank, itmp, n);
  if (thread_rank < n) {
    particle_mover_t pm = args.temp_pm_array[itmp + thread_rank];
    particle_t p = args.p0[pm.i];

    if (move_p_gpu(&p, &pm,
                   args.a0, args.g_neighbor,
                   args.g_rangel, args.g_rangeh, qsp))  // Unlikely
    {
      // assume max_nm is large enough
      int the = atomicAdd(nm, 1);
      args.pm_array[the] = pm;
    }

    args.p0[pm.i] = p;
  }
}

__global__ void advance_p_gpu(advance_p_gpu_args args) {
  const int block_rank = blockIdx.x;
  const int n_block = gridDim.x;
  const int thread_rank = threadIdx.x;
  const int n_thread = blockDim.x;
  const int stride_size = args.stride_size;

  const float qdt_2mc = args.qdt_2mc;
  const float cdt_dx = args.cdt_dx;
  const float cdt_dy = args.cdt_dy;
  const float cdt_dz = args.cdt_dz;
  const float qsp = args.qsp;
  const float one = 1.0;
  const float one_third = 1.0 / 3.0;
  const float two_fifteenths = 2.0 / 15.0;

  float dx, dy, dz, ux, uy, uz, q;
  float hax, hay, haz, cbx, cby, cbz;
  float v0, v1, v2, v3, v4, v5;
  int itmp, n;
  int *nm = args.nm;

  GPU_DISTRIBUTE(args.np, stride_size, block_rank, itmp, n);
  particle_t *p_global = args.p0;
  accumulator_t *a_global = args.a0;
  particle_mover_t *pm_array_global = args.temp_pm_array;
  particle_mover_t pm;
  const interpolator_t *f_global = args.f0;
  int prev_i = -1;

  interpolator_t f;
  for (int block_i = itmp; block_i < itmp + n; block_i += n_thread) {
    if (block_i + thread_rank < itmp + n) {
      particle_t p = p_global[block_i + thread_rank];
      f = f_global[p.i];

      dx = p.dx;  // Load position
      dy = p.dy;
      dz = p.dz;

      hax = qdt_2mc * ((f.ex + dy * f.dexdy) + dz * (f.dexdz + dy * f.d2exdydz));
      hay = qdt_2mc * ((f.ey + dz * f.deydz) + dx * (f.deydx + dz * f.d2eydzdx));
      haz = qdt_2mc * ((f.ez + dx * f.dezdx) + dy * (f.dezdy + dx * f.d2ezdxdy));

      cbx = f.cbx + dx * f.dcbxdx;  // Interpolate B
      cby = f.cby + dy * f.dcbydy;
      cbz = f.cbz + dz * f.dcbzdz;

      ux = p.ux;  // Load momentum
      uy = p.uy;
      uz = p.uz;
      q = p.w;

      ux += hax;  // Half advance E
      uy += hay;
      uz += haz;

      v0 = qdt_2mc * rsqrtf(one + (ux * ux + (uy * uy + uz * uz)));

      // Boris - scalars
      v1 = cbx * cbx + (cby * cby + cbz * cbz);
      v2 = (v0 * v0) * v1;
      v3 = v0 * (one + v2 * (one_third + v2 * two_fifteenths));
      v4 = v3 / (one + v1 * (v3 * v3));
      v4 += v4;

      v0 = ux + v3 * (uy * cbz - uz * cby);  // Boris - uprime
      v1 = uy + v3 * (uz * cbx - ux * cbz);
      v2 = uz + v3 * (ux * cby - uy * cbx);

      ux += v4 * (v1 * cbz - v2 * cby);  // Boris - rotation
      uy += v4 * (v2 * cbx - v0 * cbz);
      uz += v4 * (v0 * cby - v1 * cbx);

      ux += hax;  // Half advance E
      uy += hay;
      uz += haz;

      p.ux = ux;  // Store momentum
      p.uy = uy;
      p.uz = uz;

      v0 = rsqrtf(one + (ux * ux + (uy * uy + uz * uz)));

      // Get norm displacement

      ux *= cdt_dx;
      uy *= cdt_dy;
      uz *= cdt_dz;

      ux *= v0;
      uy *= v0;
      uz *= v0;

      v0 = dx + ux;  // Streak midpoint (inbnds)
      v1 = dy + uy;
      v2 = dz + uz;

      v3 = v0 + ux;  // New position
      v4 = v1 + uy;
      v5 = v2 + uz;

      // FIXME-KJB: COULD SHORT CIRCUIT ACCUMULATION IN THE CASE WHERE QSP==0!
      if (v3 <= one && v4 <= one && v5 <= one &&  // Check if inbnds
          -v3 <= one && -v4 <= one && -v5 <= one) {
        // Common case (inbnds).  Note: accumulator values are 4 times
        // the total physical charge that passed through the appropriate
        // current quadrant in a time-step.

        q *= qsp;

        p.dx = v3;  // Store new position
        p.dy = v4;
        p.dz = v5;

        dx = v0;  // Streak midpoint
        dy = v1;
        dz = v2;

        v5 = q * ux * uy * uz * one_third;  // Compute correction

        float *a = (float *)(a_global + p.i);  // Get accumulator

#define ACCUMULATE_J(X, Y, Z, offset)                         \
  v4 = q * u##X;   /* v2 = q ux                            */ \
  v1 = v4 * d##Y;  /* v1 = q ux dy                         */ \
  v0 = v4 - v1;    /* v0 = q ux (1-dy)                     */ \
  v1 += v4;        /* v1 = q ux (1+dy)                     */ \
  v4 = one + d##Z; /* v4 = 1+dz                            */ \
  v2 = v0 * v4;    /* v2 = q ux (1-dy)(1+dz)               */ \
  v3 = v1 * v4;    /* v3 = q ux (1+dy)(1+dz)               */ \
  v4 = one - d##Z; /* v4 = 1-dz                            */ \
  v0 *= v4;        /* v0 = q ux (1-dy)(1-dz)               */ \
  v1 *= v4;        /* v1 = q ux (1+dy)(1-dz)               */ \
  v0 += v5;        /* v0 = q ux [ (1-dy)(1-dz) + uy*uz/3 ] */ \
  v1 -= v5;        /* v1 = q ux [ (1+dy)(1-dz) - uy*uz/3 ] */ \
  v2 -= v5;        /* v2 = q ux [ (1-dy)(1+dz) - uy*uz/3 ] */ \
  v3 += v5;        /* v3 = q ux [ (1+dy)(1+dz) + uy*uz/3 ] */ \
  atomicAdd(a + offset + 0, v0);                              \
  atomicAdd(a + offset + 1, v1);                              \
  atomicAdd(a + offset + 2, v2);                              \
  atomicAdd(a + offset + 3, v3);

        ACCUMULATE_J(x, y, z, 0);
        ACCUMULATE_J(y, z, x, 4);
        ACCUMULATE_J(z, x, y, 8);

#undef ACCUMULATE_J

      } else {
        pm.dispx = ux;
        pm.dispy = uy;
        pm.dispz = uz;

        pm.i = block_i + thread_rank;

        int the = atomicAdd(nm, 1);
        pm_array_global[the] = pm;
      }

      p_global[block_i + thread_rank] = p;
    }
  }

}
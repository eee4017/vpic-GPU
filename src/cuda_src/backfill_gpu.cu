#include "hip/hip_runtime.h"
#include "gpu.cuh"
#include "backfill_gpu.cuh"
#include "gpu_util.cuh"
#include <hip/hip_cooperative_groups.h>

__global__
void back_fill_stage_1(particle_t* device_p, particle_mover_t* device_pm, 
               particle_t* device_particle_temp, int *device_particle_counter,
               particle_t* particle_selected, int np, int nm, const int block_size){
  const int block_rank = blockIdx.x;
  const int n_block = gridDim.x;
  const int thread_rank = threadIdx.x;
  const int n_thread = blockDim.x;

  int itmp, n;
  GPU_DISTRIBUTE(nm, block_size, block_rank, itmp, n);
  particle_mover_t pm;
  particle_t p, p_backfill;
  if(thread_rank < n){
    pm = device_pm[itmp + thread_rank];
    p = device_p[pm.i];
    particle_selected[itmp + thread_rank] = p;
    device_p[pm.i].i = -31;
  }
}

__global__
void back_fill_stage_2(particle_t* device_p, particle_mover_t* device_pm, 
               particle_t* device_particle_temp, int *device_particle_counter,
               particle_t* particle_selected, int np, int nm, const int block_size){

  const int block_rank = blockIdx.x;
  const int n_block = gridDim.x;
  const int thread_rank = threadIdx.x;
  const int n_thread = blockDim.x;

  int itmp, n;
  GPU_DISTRIBUTE(nm, block_size, block_rank, itmp, n);
  
  particle_mover_t pm;
  particle_t p, p_backfill;
  if(thread_rank < n){
    pm = device_pm[itmp + thread_rank];
    p_backfill = device_p[np - nm + (itmp + thread_rank)];
    if(p_backfill.i != -31){
      if (pm.i < np - nm){
        device_p[pm.i] = p_backfill;
      } else {
        int idx = atomicAdd(device_particle_counter, 1);
        device_particle_temp[idx] = p_backfill;
      } 
    }
  }
}

__global__
void back_fill_stage_3(particle_t* device_p, particle_mover_t* device_pm, 
               particle_t* device_particle_temp, int *device_particle_counter,
               particle_t* particle_selected, int np, int nm, const int block_size){

  const int block_rank = blockIdx.x;
  const int n_block = gridDim.x;
  const int thread_rank = threadIdx.x;
  const int n_thread = blockDim.x;

  int itmp, n;
  GPU_DISTRIBUTE(nm, block_size, block_rank, itmp, n);

  particle_mover_t pm;
  particle_t p;
  if(thread_rank < n){
    pm = device_pm[itmp + thread_rank];
    if(pm.i < np - nm){
      p = device_p[pm.i];
      if(p.i == -31){
        int idx = atomicSub(device_particle_counter, 1); 
        device_p[pm.i] = device_particle_temp[idx - 1];
      }
    }
    // pm.i = itmp + thread_rank;
    // device_pm[itmp + thread_rank] = pm;
  }
  __syncthreads();

}

__global__
void findPAndPm(particle_t * device_p, particle_mover_t * device_pm, 
                particle_t * d_p0,  int np, int nm)
{
  int tid = blockIdx.x *blockDim.x + threadIdx.x;

  if(tid == 0){
    particle_t *         p = device_p;
    particle_mover_t *  pm = device_pm + nm - 1;

    for( ; nm; pm--, nm-- ) {
        int i = pm->i;

        //copy p/pm
        d_p0[nm-1] = p[i];
        // d_pm[nm-1] = *pm;

        np--;
        p[i] = p[np];
      }
  }
}

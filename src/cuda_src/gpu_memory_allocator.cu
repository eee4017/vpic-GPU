#include <cassert>
#include <map>

#include "gpu.cuh"
#include "gpu_util.cuh"
using namespace vpic_gpu;
using namespace std;

gpu_memory_allocator gm;

device_pointer gpu_memory_allocator::map_to_device(host_pointer ptr, size_t size = 0) {
  auto it = host_device_map.find(ptr);
  if (it != host_device_map.end()) {
    return it->second;
  }
  assert(size != 0);
  return copy_to_device(ptr, size);
}

device_pointer gpu_memory_allocator::copy_to_device(host_pointer ptr, size_t size) {
  device_pointer dev_ptr;
  auto it = host_device_map.find(ptr);
  if (it == host_device_map.end()) {
    gpuErrchk(hipMalloc(&dev_ptr, size));
    device_array_size.insert(make_pair(ptr, size));
    host_device_map.insert(make_pair(ptr, dev_ptr));
  } else {
    dev_ptr = it->second;
  }
  if (device_array_size[dev_ptr] > size) {
    ERROR(("gpu_memory_allocator error, array size %d is larger than the original array", size));
  }
  gpuErrchk(hipMemcpy(dev_ptr, ptr, size, hipMemcpyHostToDevice));
  return dev_ptr;
}

void gpu_memory_allocator::realloc(host_pointer ptr, size_t original_size, size_t new_size) {
  auto it = host_device_map.find(ptr);
  assert(it != host_device_map.end());
  device_pointer original_array = it->second;
  device_pointer new_array;
  gpuErrchk(hipMalloc(&new_array, new_size));
  gpuErrchk(hipMemcpy(new_array, original_array, original_size, hipMemcpyDeviceToDevice));
  gpuErrchk(hipFree(original_array));
}

void gpu_memory_allocator::copy_to_host(host_pointer ptr, size_t size) {
  auto it = host_device_map.find(ptr);
  assert(it != host_device_map.end());
  device_pointer dev_ptr = it->second;
  gpuErrchk(hipMemcpy(ptr, dev_ptr, size, hipMemcpyDeviceToHost));
}

/* with stream version */

device_pointer gpu_memory_allocator::map_to_device(host_pointer ptr, hipStream_t stream, size_t size = 0) {
  auto it = host_device_map.find(ptr);
  if (it != host_device_map.end()) {
    return it->second;
  }
  assert(size != 0);
  return copy_to_device(ptr, size, stream);
}

device_pointer gpu_memory_allocator::copy_to_device(host_pointer ptr, size_t size, hipStream_t stream) {
  device_pointer dev_ptr;
  auto it = host_device_map.find(ptr);
  if (it == host_device_map.end()) {
    gpuErrchk(hipMalloc(&dev_ptr, size));
    device_array_size.insert(make_pair(ptr, size));
    host_device_map.insert(make_pair(ptr, dev_ptr));
  } else {
    dev_ptr = it->second;
  }
  if (device_array_size[dev_ptr] > size) {
    ERROR(("gpu_memory_allocator error, array size %d is larger than the original array", size));
  }
  gpuErrchk(hipMemcpyAsync(dev_ptr, ptr, size, hipMemcpyHostToDevice, stream));
  return dev_ptr;
}

void gpu_memory_allocator::realloc(host_pointer ptr, size_t original_size, size_t new_size, hipStream_t stream) {
  auto it = host_device_map.find(ptr);
  assert(it != host_device_map.end());
  device_pointer original_array = it->second;
  device_pointer new_array;
  gpuErrchk(hipMalloc(&new_array, new_size));
  gpuErrchk(hipMemcpyAsync(new_array, original_array, original_size, hipMemcpyDeviceToDevice, stream));
  gpuErrchk(hipFree(original_array));
}

void gpu_memory_allocator::copy_to_host(host_pointer ptr, size_t size, hipStream_t stream) {
  auto it = host_device_map.find(ptr);
  assert(it != host_device_map.end());
  device_pointer dev_ptr = it->second;
  gpuErrchk(hipMemcpyAsync(ptr, dev_ptr, size, hipMemcpyDeviceToHost, stream));
}
